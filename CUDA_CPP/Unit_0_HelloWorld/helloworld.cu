
#include <hip/hip_runtime.h>
#include<stdio.h> // Khai báo thư viện chuẩn cho xuất kết quả ra màn hình

// Khai báo hàm được chạy trên GPU.
// Hàm có thể được gọi từ hàm main
__global__ void cuda_hello(){ 
    printf("Hello World from GPU!\n");
}

// Hàm main, còn được gọi là "Kernels"
int main() {
    cuda_hello<<<1,10>>>(); // kernel launch - Khởi chạy hàm trên GPU với tham số (1. 10)
    hipDeviceSynchronize(); // Chặn cho tới khi các lệnh trên hoàn thành.
    return 0;
}